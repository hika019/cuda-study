#include "hip/hip_runtime.h"
#include <iostream>

const int  braket_size = 2;

struct Braket{
    double vector[braket_size];
    bool ket;


};

__global__ void gen(){
    Braket a = Braket();
    a.vector[0] = (double)rand()/RAND_MAX;
}

int main(){
    gen<<<1000, 1000>>>();

    return 0;
}